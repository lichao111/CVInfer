#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "timer_gpu.cuh"
#include "tools/logger.h"

namespace cv_infer
{
TimerGPU::TimerGPU()
{
    checkRuntime(hipEventCreate((hipEvent_t *)&Start));
    checkRuntime(hipEventCreate((hipEvent_t *)&Stop));
}

TimerGPU::~TimerGPU()
{
    checkRuntime(hipEventDestroy((hipEvent_t)Start));
    checkRuntime(hipEventDestroy((hipEvent_t)Stop));
}

void TimerGPU::start(void *stream)
{
    Stream = stream;
    checkRuntime(hipEventRecord((hipEvent_t)Start, (hipStream_t)stream));
}

float TimerGPU::stop(const char *prefix, bool print)
{
    checkRuntime(hipEventRecord((hipEvent_t)Stop, (hipStream_t)Stream));
    checkRuntime(hipEventSynchronize((hipEvent_t)Stop));

    float latency = 0;
    checkRuntime(hipEventElapsedTime(&latency, (hipEvent_t)Start, (hipEvent_t)Stop));

    if (print)
    {
        LOGI("[%s]: %.5f ms", prefix, latency);
    }
    return latency;
}
}  // namespace cv_infer