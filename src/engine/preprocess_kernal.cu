#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "preprocess_kernal.cuh"
#include "timer_gpu.cuh"

using namespace cv_infer;
namespace CUDAKernal
{
__global__ void ConverHWC2CHWMeanStdKernel(const unsigned char* src, int h, int w, int c, const float* mean,
                                           const float* scale, float* dst)
{
    // from copilot :
    /*
    在 CUDA 编程中，计算线程索引的方式有多种，选择哪种方式取决于具体的应用场景和需求。以下是对两种索引计算方式的解释：
    方式一：直接计算三维索引
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        int z = blockIdx.z * blockDim.z + threadIdx.z;

    方式二：计算一维索引并转换为三维索引
        int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
        int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) +
(threadIdx.y * blockDim.x) + threadIdx.x;
        int x = threadId % width;
        int y = (threadId / width) % height;
        int z = threadId / (width * height);
    */

    /*  HWC->CHW得转换原理
        HWC: BGRBGRBGRBGR......BGRBGRBGR  (行优先)
        CHW: BBB...BBBGGG...GGGRRR...RRR
        source_index = (x, y, z)
        则source_index对应得是第 y*w*c + x*c + z 个元素
        其转换后得目标位置为 z*h*w + y*w + x
    */

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < w && y < h && z < c)
    {
        dst[z * h * w + y * w + x] = ((float)(src[y * w * c + x * c + z] - mean[z])) / scale[z];
    }
}

// 函数功能：
// 1. 将HWC格式的图像转换为CHW格式的图像
// 2. 进行alpha,beta归一化
// 3. 进行保持比例缩放, 使用固定值padding
__global__ void ConverHWC2CHWAlpahNormResizeKeepRatioKernel(const unsigned char* src, int src_h, int src_w, int src_c,
                                                            int dst_h, int dst_w, int dst_c, float alpha, float beta,
                                                            int fill_value, int stride, float* dst)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    // dst_index = (x,y,z)
    if (x < dst_w && y < dst_h && z < src_c)
    {
        // 计算缩放比例
        float scale = min((float)dst_w / src_w, (float)dst_h / src_h);
        int   new_w = (int)(src_w * scale);
        int   new_h = (int)(src_h * scale);

        // 计算填充
        int pad_x = (dst_w - new_w) / 2;
        int pad_y = (dst_h - new_h) / 2;

        // 检查是否在填充区域
        if (x < pad_x || x >= pad_x + new_w || y < pad_y || y >= pad_y + new_h)
        {
            // 填充颜色
            dst[z * dst_h * dst_w + y * dst_w + x] = fill_value * alpha + beta;
            // CHW格式 所以是z*h*w + y*w + x ，否则为y*w*c + x*c + z
        }
        else
        {
            // 计算源图像中的坐标
            int src_x = (int)((x - pad_x) / scale);
            int src_y = (int)((y - pad_y) / scale);
            // 将 BGR 转换为 RGB
            int rgb_index;
            if (z == 0)
            {
                rgb_index = src_y * src_w * src_c + src_x * src_c + 2;  // B -> R
            }
            else if (z == 1)
            {
                rgb_index = src_y * src_w * src_c + src_x * src_c + 1;  // G -> G
            }
            else
            {
                rgb_index = src_y * src_w * src_c + src_x * src_c + 0;  // R -> B
            }

            // 进行 HWC 到 CHW 的转换并归一化
            dst[z * dst_h * dst_w + y * dst_w + x] = ((float)(src[rgb_index])) * alpha + beta;
        }
    }
}

void ConverHWC2CHWMeanStd(const unsigned char* src, int h, int w, int c, const float* mean, const float* scale,
                          float* dst)
{
    // int jobs = h * w;
    // auto grid = grid_dims(jobs);
    // auto block = block_dims(jobs);
    dim3 block(16, 16, 1);
    auto grid_x = (w + block.x - 1) / block.x;
    auto grid_y = (h + block.y - 1) / block.y;
    auto grid_z = (c + block.z - 1) / block.z;
    dim3 grid(grid_x, grid_y, grid_z);

    // stream
    hipStream_t stream;
    hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, 0);
    TimerGPU timer;
    timer.start(stream);

    // copy src to device
    unsigned char* src_d;
    hipMallocAsync(&src_d, h * w * c * sizeof(unsigned char), stream);
    hipMemcpyAsync(src_d, src, h * w * c * sizeof(unsigned char), hipMemcpyHostToDevice, stream);
    // copy mean to device
    float* mean_d;
    hipMallocAsync(&mean_d, 3 * sizeof(float), stream);
    hipMemcpyAsync(mean_d, mean, c * sizeof(float), hipMemcpyHostToDevice, stream);
    // copy scale to device
    float* scale_d;
    hipMallocAsync(&scale_d, 3 * sizeof(float), stream);
    hipMemcpyAsync(scale_d, scale, c * sizeof(float), hipMemcpyHostToDevice, stream);

    ConverHWC2CHWMeanStdKernel<<<grid, block, 0, stream>>>(src_d, h, w, c, mean_d, scale_d, dst);

    // realize async
    hipStreamSynchronize(stream);
    timer.stop("ConverHWC2CHWMeanStd", true);

    hipFree(src_d);
    hipFree(mean_d);
    hipFree(scale_d);
    hipStreamDestroy(stream);
}

void ConverHWC2CHWAlpahNormResizeKeepRatio(const unsigned char* src, int src_h, int src_w, int src_c, int dst_h,
                                           int dst_w, int dst_c, float alpha, float beta, float fill_value, int stride,
                                           float* dst)
{
    dim3 block(16, 16, 1);
    auto grid_x = (dst_w + block.x - 1) / block.x;
    auto grid_y = (dst_h + block.y - 1) / block.y;
    auto grid_z = (dst_c + block.z - 1) / block.z;
    dim3 grid(grid_x, grid_y, grid_z);

    // stream
    hipStream_t stream;
    hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, 0);
    TimerGPU timer;
    timer.start(stream);

    // copy src to device
    unsigned char* src_d;
    hipMallocAsync(&src_d, src_h * src_w * src_c * sizeof(unsigned char), stream);
    hipMemcpyAsync(src_d, src, src_h * src_w * src_c * sizeof(unsigned char), hipMemcpyHostToDevice, stream);

    ConverHWC2CHWAlpahNormResizeKeepRatioKernel<<<grid, block, 0, stream>>>(
        src_d, src_h, src_w, src_c, dst_h, dst_w, dst_c, alpha, beta, fill_value, stride, dst);

    // realize async
    hipStreamSynchronize(stream);
    timer.stop("ConverHWC2CHWNorm", true);

    hipFree(src_d);
    hipStreamDestroy(stream);
}

}  // namespace CUDAKernal