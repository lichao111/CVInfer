#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "preprocess_kernal.cuh"
#include "timer_gpu.cuh"

using namespace cv_infer;
namespace CUDAKernal
{
__global__ void ConverHWC2CHWMeanStdKernel(const unsigned char* src, int h, int w, int c, const float* mean,
                                           const float* scale, float* dst)
{
    // from copilot :
    /*
    在 CUDA 编程中，计算线程索引的方式有多种，选择哪种方式取决于具体的应用场景和需求。以下是对两种索引计算方式的解释：
    方式一：直接计算三维索引
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        int z = blockIdx.z * blockDim.z + threadIdx.z;

    方式二：计算一维索引并转换为三维索引
        int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
        int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) +
(threadIdx.y * blockDim.x) + threadIdx.x;
        int x = threadId % width;
        int y = (threadId / width) % height;
        int z = threadId / (width * height);
    */

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < w && y < h && z < c)
    {
        dst[z * h * w + y * w + x] = ((float)(src[y * w * c + x * c + z] - mean[z])) / scale[z];
    }
}

__global__ void ConverHWC2CHWNormKernel(const unsigned char* src, int h, int w, int c, float* dst)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < w && y < h && z < c)
    {
        dst[z * h * w + y * w + x] = ((float)(src[y * w * c + x * c + z])) / 255.0;
    }
}

void ConverHWC2CHWMeanStd(const unsigned char* src, int h, int w, int c, const float* mean, const float* scale,
                          float* dst)
{
    // int jobs = h * w;
    // auto grid = grid_dims(jobs);
    // auto block = block_dims(jobs);
    dim3 block(16, 16, 1);
    auto grid_x = (w + block.x - 1) / block.x;
    auto grid_y = (h + block.y - 1) / block.y;
    auto grid_z = (c + block.z - 1) / block.z;
    dim3 grid(grid_x, grid_y, grid_z);

    // stream
    hipStream_t stream;
    hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, 0);
    TimerGPU timer;
    timer.start(stream);

    // copy src to device
    unsigned char* src_d;
    hipMallocAsync(&src_d, h * w * c * sizeof(unsigned char), stream);
    hipMemcpyAsync(src_d, src, h * w * c * sizeof(unsigned char), hipMemcpyHostToDevice, stream);
    // copy mean to device
    float* mean_d;
    hipMallocAsync(&mean_d, 3 * sizeof(float), stream);
    hipMemcpyAsync(mean_d, mean, c * sizeof(float), hipMemcpyHostToDevice, stream);
    // copy scale to device
    float* scale_d;
    hipMallocAsync(&scale_d, 3 * sizeof(float), stream);
    hipMemcpyAsync(scale_d, scale, c * sizeof(float), hipMemcpyHostToDevice, stream);

    ConverHWC2CHWMeanStdKernel<<<grid, block, 0, stream>>>(src_d, h, w, c, mean_d, scale_d, dst);

    // realize async
    hipStreamSynchronize(stream);
    timer.stop("ConverHWC2CHWMeanStd", true);

    hipFree(src_d);
    hipFree(mean_d);
    hipFree(scale_d);
    hipStreamDestroy(stream);
}

void ConverHWC2CHWNorm(const unsigned char* src, int h, int w, int c, float* dst)
{
    dim3 block(16, 16, 1);
    auto grid_x = (w + block.x - 1) / block.x;
    auto grid_y = (h + block.y - 1) / block.y;
    auto grid_z = (c + block.z - 1) / block.z;
    dim3 grid(grid_x, grid_y, grid_z);

    // stream
    hipStream_t stream;
    hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, 0);
    TimerGPU timer;
    timer.start(stream);

    // copy src to device
    unsigned char* src_d;
    hipMallocAsync(&src_d, h * w * c * sizeof(unsigned char), stream);
    hipMemcpyAsync(src_d, src, h * w * c * sizeof(unsigned char), hipMemcpyHostToDevice, stream);

    ConverHWC2CHWNormKernel<<<grid, block, 0, stream>>>(src_d, h, w, c, dst);

    // realize async
    hipStreamSynchronize(stream);
    timer.stop("ConverHWC2CHWNorm", true);

    hipFree(src_d);
    hipStreamDestroy(stream);
}

}  // namespace CUDAKernal